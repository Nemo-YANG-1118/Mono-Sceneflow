#include "hip/hip_runtime.h"

#include <hip/hip_runtime.h>
#include <>

#include <iostream>


namespace stereoscene{


	__global__
	static void
	approximateMotionField_L2_Kernel(
	float* const d_dval , 
	float* const d_x3 , 
	const unsigned char* const d_dmask , 
	const float* const d_ray3 ,
	const float* const d_du , 
	const float* const d_dv ,
	const float* const d_Pvec , 
	const float* const d_Cvec ,
	const unsigned int NPoints , 
	const unsigned int NPad , 
	const unsigned int NWidth , 
	const unsigned int NHeight) {
		
		const unsigned int x3Ind = blockDim.x*blockIdx.x + threadIdx.x ;
		
		if(x3Ind< NPoints) {

			const unsigned int nw = x3Ind%NWidth , nh = x3Ind/NWidth ;

			const unsigned char* const pdmask = d_dmask+x3Ind ;

			if(*pdmask) {

				float *const px3 = d_x3 + x3Ind*3 ,
					*const pdval = d_dval + x3Ind ;

				const float *const pray3 = d_ray3 + x3Ind*3 ;

				float sx3Mot(0) , x2Mot(0) ;

				for(unsigned int npd=0 ; npd< NPad*2 ; npd++) {

					const float *const Pvec = d_Pvec+npd*12 ;


						///step.1: calculate jacobian of each x3 point
						///
						const float xh = Pvec[0]*px3[0]+Pvec[1]*px3[1]+Pvec[2]*px3[2]+Pvec[3],
							yh = Pvec[4]*px3[0]+Pvec[5]*px3[1]+Pvec[6]*px3[2]+Pvec[7] ,
							zh = Pvec[8]*px3[0]+Pvec[9]*px3[1]+Pvec[10]*px3[2]+Pvec[11] ,
							zh2 = zh*zh ;

						const float Jux = (Pvec[0]*zh-xh*Pvec[8])/zh2 , 
							Juy = (Pvec[1]*zh-xh*Pvec[9])/zh2 , 
							Juz = (Pvec[2]*zh-xh-Pvec[10])/zh2 ,
							Jvx = (Pvec[4]*zh-yh*Pvec[8])/zh2 , 
							Jvy = (Pvec[5]*zh-yh*Pvec[9])/zh2 ,
							Jvz = (Pvec[6]*zh-yh*Pvec[10])/zh2 ;


						///step.2: calculate 3d ray and 2d directionary vector
						///
						const float x2uDir = Jux*pray3[0]+Juy*pray3[1]+Juz*pray3[2] ;
						const float x2vDir = Jvx*pray3[0]+Jvy*pray3[1]+Jvz*pray3[2] ;
 

						///step.3: calculate x3's motion scale
						///
						const float *const pdu = d_du + (NHeight*NWidth)*npd + nh*NWidth+nw,
							*const pdv = d_dv + (NHeight*NWidth)*npd + nh*NWidth+nw ;

						sx3Mot += (x2uDir*(*pdu) + x2vDir*(*pdv)) ;
						x2Mot += (x2uDir*x2uDir + x2vDir*x2vDir) ;
				}

				sx3Mot /= x2Mot ;


				///step.4: update x3's position and depth value by using motion variation
				///
				px3[0] += sx3Mot*pray3[0] , px3[1] += sx3Mot*pray3[1] , px3[2] += sx3Mot*pray3[2] ;

				pdval[0] = sqrtf((px3[0]-d_Cvec[0])*(px3[0]-d_Cvec[0]) + 
					(px3[1]-d_Cvec[1])*(px3[1]-d_Cvec[1]) + 
					(px3[2]-d_Cvec[2])*(px3[2]-d_Cvec[2]));

			}
		}

	}


}


#include <opencv2/imgproc.hpp>
#include <opencv2/highgui.hpp>
#include <opencv2/cuda.hpp>

#include "SceneFlowOptimizer.h"


#ifdef _SEE_IMAGE_
#undef _SEE_IMAGE_
#endif
#define _SEE_IMAGE_(img_ , wname_)\
	cv::namedWindow(wname_ , 0);\
	if(img_.cols<1680 && img_.rows<1050)\
	cv::resizeWindow(wname_ , img_.cols , img_.rows);\
		else\
	cv::resizeWindow(wname_ , img_.cols/2 , img_.rows/2);\
	cv::imshow(wname_ , img_);\
	cv::waitKey(0);\
	cv::destroyWindow(wname_);


#undef _PRINT_VEC_
#define _PRINT_VEC_(_biter , _eiter , _typename)\
	fflush(stdout);\
	std::copy(_biter , _eiter , (std::ostream_iterator<_typename>(std::cout , "\t\t"))) ;\
	std::cout<<std::endl ;\
	fflush(stdout);


#undef _MAX_ITERS_
#define _MAX_ITERS_ 1


namespace stereoscene{


	bool
		SceneFlowOptimizer::
		approximateMotionField_CUDA() {


		const cv::Size& MapSize = _DepthMap_.size() ;

		
		const unsigned int NPoints = _vec_x3Points_.size()/3,
			NCams = _vec_pPCams_.size(),
			NPad = NCams/2,
			KthRefPCam = NCams/2 ,
			NWidth = MapSize.width ,
			NHeight = MapSize.height ;


		const std::vector<float>& Cvec = _vec_pPCams_[KthRefPCam]->_Cvec_ ;


		///step.1: import data from host to device
		///
		float *d_x3(NULL) , *d_ray3(NULL) , *d_Cvec(NULL) , *d_dval(NULL) ;

		hipMalloc( &d_x3 , sizeof(float)*NPoints*3 ) ;
		hipMemcpyAsync(d_x3,  _vec_x3Points_.data() , sizeof(float)*NPoints*3 , hipMemcpyHostToDevice) ;

		hipMalloc( &d_ray3 , sizeof(float)*NPoints*3 ) ;
		hipMemcpyAsync(d_ray3 , _vec_x3Rays_.data() , sizeof(float)*NPoints*3 , hipMemcpyHostToDevice) ;

		hipMalloc( &d_Cvec , sizeof(float)*3 ) ;
		hipMemcpyAsync(d_Cvec , Cvec.data() , sizeof(float)*3 , hipMemcpyHostToDevice) ;

		hipMalloc( &d_dval , sizeof(float)*NWidth*NHeight ) ;
		hipMemcpyAsync(d_dval , _DepthMap_.ptr<float>(0) , sizeof(float)*NWidth*NHeight , hipMemcpyHostToDevice);



		unsigned char *d_dmask(NULL);

		hipMalloc(&d_dmask,sizeof(unsigned char)*NWidth*NHeight) ;
		hipMemcpyAsync(d_dmask,_DepthMask_.ptr<unsigned char>(0),sizeof(unsigned char)*NWidth*NHeight,hipMemcpyHostToDevice) ;


		float *d_du(NULL) , *d_dv(NULL) , *d_Pvec(NULL);

		hipMalloc( &d_Pvec , sizeof(float)*12*2*NPad) ;
		hipMalloc( &d_du , sizeof(float)*(NWidth*NHeight)*2*NPad ) ;
		hipMalloc( &d_dv , sizeof(float)*(NWidth*NHeight)*2*NPad ) ;


		std::map<int,cv::Mat>::const_iterator it_uFlow=_map_uFlow_.begin(), it_vflow=_map_vFlow_.begin() ;

		for(int cntPCam=0 , cntPad=0 ; cntPCam<NCams ; cntPCam++) {
			if(cntPCam != KthRefPCam){

				hipMemcpyAsync(d_du + cntPad*NWidth*NHeight , it_uFlow->second.ptr<float>(0),sizeof(float)*NWidth*NHeight,hipMemcpyHostToDevice) ;
				hipMemcpyAsync(d_dv + cntPad*NWidth*NHeight , it_vflow->second.ptr<float>(0),sizeof(float)*NWidth*NHeight,hipMemcpyHostToDevice) ;

				hipMemcpyAsync(d_Pvec + cntPad*12 , _vec_pPCams_[cntPCam]->_P_.data(),sizeof(float)*12,hipMemcpyHostToDevice) ;

				//std::cout<<it_uFlow->second.cols<<"\t"<<it_uFlow->second.rows<<"\n";

				++it_uFlow,++it_vflow;
	
				cntPad++ ;
				//_PRINT_VEC_(_vec_pPCams_[cntPCam]->_P_.data() ,_vec_pPCams_[cntPCam]->_P_.data()+12 , float);
		
				//std::cout<<cntPCam<<"\n";
			}
		}


		//hipError_t cuError0 = hipGetLastError() ;
		//if(cuError0 != hipSuccess) {
		//	fflush(stderr) ;
		//	fprintf(stderr,"\nError: Error happened in < MotionFieldApproximator.cu / cuError0 >\n") ;
		//	fflush(stderr) ;
		//	exit(0);
		//}


		///step.2: implement monocular scene flow
		///
		const unsigned int NThreads = 256 ,
			NBlocks = (NPoints+NThreads-1)/NThreads ;

		for(int niter=0 ; niter<_MAX_ITERS_ ; niter++) {
			approximateMotionField_L2_Kernel<<< NBlocks , NThreads >>>( 
				d_dval , d_x3 , d_dmask , d_ray3 , d_du , d_dv  , d_Pvec , d_Cvec ,
				NPoints , NPad , NWidth , NHeight) ;
		}


		hipError_t cuError1 = hipDeviceSynchronize() ;
		if(cuError1 != hipSuccess){
			fflush(stderr) ;
			fprintf(stderr , "\nError: Error happened in < MotionFieldApproximatror.cu / cuError1>\n");
			fflush(stderr) ;
			exit(0);
		}


		///step.3: export data from device to host
		///
		cv::Mat tmpDepth(_DepthMap_.size() , CV_32FC1);
		hipMemcpy(tmpDepth.ptr<float>(0) , d_dval , sizeof(float)*NPoints , hipMemcpyDeviceToHost) ;

		hipFree(d_du) ;
		hipFree(d_dv) ;
		hipFree(d_ray3) ;
		hipFree(d_x3) ;
		hipFree(d_Pvec);
		hipFree(d_Cvec) ;
		hipFree(d_dval) ;
		hipFree(d_dmask);



		cv::bilateralFilter(tmpDepth , _DepthMap_ , 5 , 0.1 , 5);


		return true ;
	}


}